#include "hip/hip_runtime.h"
//
// Created by ilya on 28.02.2021.
//

#include "Physics.hpp"
#include <algorithm>
#include <limits>
#include <iostream>

constexpr int THREAD_WARP_SIZE = 32;

__device__ bool intersect(const Body &l, const Body &r) {
    float X = l.x - r.x, Y = l.y - r.y, R = r.radius + l.radius;
    return (X * X + Y * Y - R * R) < 0.f;
}

__device__ void merge_two_bodies(Body &a, const Body &b) {
    if (a.mass == 0.f || b.mass == 0.f)
        return;

    a.x = a.radius > b.radius ? a.x : b.x;
    a.y = a.radius > b.radius ? a.y : b.y;

    a.radius = sqrt(a.radius * a.radius + b.radius * b.radius);

    auto mass_sum = b.mass + a.mass;
    a.speed_x = a.speed_x * (a.mass / mass_sum) + (b.speed_x) * (b.mass / mass_sum);
    a.speed_y = a.speed_y * (a.mass / mass_sum) + (b.speed_y) * (b.mass / mass_sum);

    a.mass = mass_sum;
}

__device__ void merge(const Body *body_vec_in, Body *body_vec_out, int bodies_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ Body cached_bodies[THREAD_WARP_SIZE];
    Body current_body = body_vec_in[idx];

    cached_bodies[threadIdx.x] = idx < bodies_count ? current_body : Body{0, 0, 0, 0, 0, 0, 0, idx, -1000};

    __syncthreads();

    for (int i = 0; i < THREAD_WARP_SIZE; i++) {
        if (threadIdx.x == i || current_body.mass == 0.f) continue;
        if (cached_bodies[i].mass == 0.f) continue;

        if (intersect(current_body, cached_bodies[i])) {
            if (threadIdx.x > i) {
                current_body = {0, 0, 0, 0, 0, 0, 0, idx, blockIdx.x * blockDim.x + i};
            } else {
                merge_two_bodies(current_body, cached_bodies[i]);
                current_body.dbg_1 = idx;
                current_body.dbg_2 = blockIdx.x * blockDim.x + i;
            }
        }
    }

    __syncthreads();

    for (int i = 0; i < blockIdx.x; i++) {
        cached_bodies[threadIdx.x] = body_vec_in[i * blockDim.x + threadIdx.x];
        __syncthreads();

        if (current_body.mass == 0.f) continue;

        for (int j = 0; j < THREAD_WARP_SIZE; j++) {
            if (cached_bodies[j].mass == 0.f)
                continue;

            if (intersect(current_body, cached_bodies[j])) {
                current_body.mass = 0.f;
            }
        }
    }

    for (int i = blockIdx.x + 1; i < gridDim.x; i++) {
        if (i * blockDim.x + threadIdx.x > bodies_count) {
            cached_bodies[threadIdx.x] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
        } else {
            cached_bodies[threadIdx.x] = body_vec_in[i * blockDim.x + threadIdx.x];
        }

        __syncthreads();

        if (current_body.mass == 0.f) continue;

        for (int j = 0; j < THREAD_WARP_SIZE; j++) {
            if (cached_bodies[j].mass == 0.f)
                continue;

            if (intersect(current_body, cached_bodies[j])) {
                merge_two_bodies(current_body, cached_bodies[j]);
            }
        }
    }

    body_vec_out[idx] = current_body;
}

__device__ void update_positions(const Body *body_vec_in, Body *body_vec_out, int bodies_count) {
    constexpr auto step = 1.f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > bodies_count)
        return;

    float Fx = 0., Fy = 0.;

    Body current_body = body_vec_out[idx];
    for (int i = 0; i < bodies_count; i++) {
        auto tmp = body_vec_in[i];
    
        if (current_body.mass == 0.f)
            continue;
        
        if (idx == i || tmp.mass == 0.f)
            continue;

        auto X = tmp.x - current_body.x, Y = tmp.y - current_body.y;
        auto D2 = (X * X + Y * Y);
        auto F = 6.674184 * 10e-9 * static_cast<double >(current_body.mass * tmp.mass) / D2;
        auto D = sqrt(D2);

        if (D < (tmp.radius + current_body.radius))
            continue;

        Fx += F * X / D;
        Fy += F * Y / D;
    }

    current_body.x += current_body.speed_x * step;
    current_body.y += current_body.speed_y * step;
    current_body.speed_x += Fx * step / current_body.mass;
    current_body.speed_y += Fy * step / current_body.mass;

    body_vec_out[idx] = current_body;
}

__global__ void update_gpu_bodies(const Body *body_vec_in, Body *body_vec_out, int bodies_count) {
    merge(body_vec_in, body_vec_out, bodies_count);
    update_positions(body_vec_in, body_vec_out, bodies_count);
}

Physics::Physics() : gpu_bodies_vec_a_(nullptr) {}

Physics::~Physics() {
    if (gpu_bodies_vec_a_){
        hipFree(gpu_bodies_vec_a_);
        hipFree(gpu_bodies_vec_b_);
    }
}

void Physics::load(std::vector<Body> &bodies) {
    if (gpu_bodies_vec_a_){
        hipFree(gpu_bodies_vec_a_);
        hipFree(gpu_bodies_vec_b_);
    }

    float min_X, min_Y, max_X, max_Y;
    max_X = max_Y = std::numeric_limits<float>::lowest();
    min_X = min_Y = std::numeric_limits<float>::max();

    for (const Body &b : bodies) {
        if (min_X > b.x)
            min_X = b.x;
        else if (max_X < b.x)
            max_X = b.x;

        if (min_Y > b.y)
            min_Y = b.y;
        else if (max_Y < b.y)
            max_Y = b.y;
    }

    std::sort(std::begin(bodies), std::end(bodies), [min_X, min_Y, max_X, max_Y](Body &a, Body &b) {
        auto diff = (b.y - a.y) * 2 + b.x - a.x;
        if (b.y < a.y) diff -= 10;
        return diff > 0;
    });

    hipMalloc(&gpu_bodies_vec_a_, sizeof(Body) * bodies.size());
    hipMalloc(&gpu_bodies_vec_b_, sizeof(Body) * bodies.size());
    // cuda Malloc Host
    hipMemcpy(gpu_bodies_vec_a_, bodies.data(), sizeof(Body) * bodies.size(), hipMemcpyHostToDevice);
}

void Physics::update(std::vector<Body> &bodies) {
    if (!bodies.empty()) {
        std::vector<Body> tmp;
        tmp.reserve(((bodies.size() / THREAD_WARP_SIZE) + 1) * THREAD_WARP_SIZE);

        for (const auto &b: bodies) {
            if (b.mass != 0.f)
                tmp.push_back(b);
        }

//        auto sz_diff = bodies.size() - tmp.size();
//        for (int i = 0; i < sz_diff; i++) {
//            tmp.push_back({});
//        }

        bodies = tmp;
    }

    dim3 threads = dim3(THREAD_WARP_SIZE, 1);
    dim3 blocks = bodies.size() / threads.x != 0 ? dim3((bodies.size() / threads.x) + 1, 1) : dim3(1, 1);

    hipMemcpy(gpu_bodies_vec_a_, bodies.data(), sizeof(Body) * bodies.size(), hipMemcpyHostToDevice);
    update_gpu_bodies<<<blocks, threads>>>(gpu_bodies_vec_a_, gpu_bodies_vec_b_, bodies.size());
    hipMemcpy(bodies.data(), gpu_bodies_vec_b_, sizeof(Body) * bodies.size(), hipMemcpyDeviceToHost);

//    float rad_sz = 0.f;
//    for (auto &a: bodies) {
//        rad_sz += a.radius;
//        if (a.dbg_1 == 0 && a.dbg_2 == 0) continue;
//        std::cout << "dbg_1: " << a.dbg_1;
//        std::cout << " | dbg_2: " << a.dbg_2 << '\n';
//        a.dbg_1 = 0;
//        a.dbg_2 = 0;
//
//    }
//    std::cout << "------------------ sz: " << bodies.size() << "------------------\n";
}
