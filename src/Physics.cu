#include "hip/hip_runtime.h"
//
// Created by ilya on 28.02.2021.
//

#include "Physics.hpp"
#include <float.h>

__device__ bool intersect(const Body &l, const Body &r) {
    float X = l.x - r.x, Y = l.y - r.y, R = r.radius + l.radius;
    return (X * X + Y * Y - R * R) < 0.f;
}

__device__ void merge(Body *body_vec, Body *result, int bodies_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    Body current_body = body_vec[idx];
    bool delete_this = false;

    if (idx > bodies_count)
        return;

    if (current_body.mass == 0.f)
        return;

    for (int i = 0; i < idx; i++) {
        if (intersect(current_body, body_vec[i])) {
            delete_this = true;
            break;
        }
    }

    if (!delete_this) {
        for (int i = idx + 1; i < bodies_count; i++) {
            auto tmp = body_vec[i];

            if (tmp.mass == 0.f)
                continue;

            if (intersect(current_body, tmp)) {
                current_body.x = current_body.radius > tmp.radius ? current_body.x : tmp.x;
                current_body.y = current_body.radius > tmp.radius ? current_body.y : tmp.y;

                current_body.radius = sqrt(current_body.radius * current_body.radius + tmp.radius * tmp.radius);

                auto mass_sum = tmp.mass + current_body.mass;
                current_body.speed_x =
                        current_body.speed_x * (current_body.mass / mass_sum) + (tmp.speed_x) * (tmp.mass / mass_sum);
                current_body.speed_y =
                        current_body.speed_y * (current_body.mass / mass_sum) + (tmp.speed_y) * (tmp.mass / mass_sum);

                current_body.mass = mass_sum;
            }
        }
    } else {
        current_body.mass = 0.f;
        current_body.radius = 0.f;
    }

    result[idx] = current_body;
}

__device__ void update_positions(Body *body_vec, Body *result, size_t bodies_count) {
    constexpr auto step = 0.2f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > bodies_count)
        return;

    float Fx = 0., Fy = 0.;

    Body &current_body = body_vec[idx];
    for (int i = 0; i < bodies_count; i++) {
        auto tmp = body_vec[i];

        if (idx == i || tmp.mass == 0.f)
            continue;

        auto X = tmp.x - current_body.x, Y = tmp.y - current_body.y;
        auto D2 = (X * X + Y * Y);
        auto F = 6.674184 * 10e-9 * (current_body.mass * tmp.mass) / D2;
        auto D = sqrt(D2);

        Fx += F * X / D;
        Fy += F * Y / D;
    }

    result[idx].x += current_body.speed_x * step;
    result[idx].y += current_body.speed_y * step;
    result[idx].speed_x += Fx * step / current_body.mass;
    result[idx].speed_y += Fy * step / current_body.mass;
}

__global__ void update_gpu_bodies(Body *body_vec, Body *result, int bodies_count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    merge(body_vec, result, bodies_count);

    __syncthreads();

    body_vec[idx] = result[idx];
    update_positions(body_vec, result, bodies_count);
}

Physics::Physics() : gpu_bodies_vec_(nullptr) {}

Physics::~Physics() {
    if (gpu_bodies_vec_)
        hipFree(gpu_bodies_vec_);
};

void Physics::load(const std::vector<Body> &bodies) {
    if (gpu_bodies_vec_)
        hipFree(gpu_bodies_vec_);

    hipMalloc(&gpu_bodies_vec_, sizeof(Body) * bodies.size());
    // cuda Malloc Host
    hipMemcpy(gpu_bodies_vec_, bodies.data(), sizeof(Body) * bodies.size(), hipMemcpyHostToDevice);
}

void Physics::update(std::vector<Body> &bodies) {
    dim3 threads = dim3(64, 1);
    dim3 blocks = bodies.size() / threads.x != 0 ? dim3((bodies.size() / threads.x) + 1, 1) : dim3(1, 1);

    // TMP
    Body *gpu_result;
    hipMalloc(&gpu_result, sizeof(Body) * bodies.size());

    hipMemcpy(gpu_result, gpu_bodies_vec_, sizeof(Body) * bodies.size(), hipMemcpyDeviceToDevice);
    update_gpu_bodies<<<blocks, threads>>>(gpu_bodies_vec_, gpu_result, bodies.size());
    hipMemcpy(bodies.data(), gpu_result, sizeof(Body) * bodies.size(), hipMemcpyDeviceToHost);
    hipMemcpy(gpu_bodies_vec_, gpu_result, sizeof(Body) * bodies.size(), hipMemcpyDeviceToDevice);

    // TMP
    hipFree(gpu_result);
}
